#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <rules.h>

#define MAX_LEN 1500

__global__ static void match(uint16_t *acArray, AcNodeGPU *contPatt, char *packets, uint16_t *packets_len, uint16_t *result)
{
	printf("GPU match\n");
	uint16_t len;
	uint16_t contId[20] = {0};
	int tmp;
	int state = 0;
	int i = 0, j = 0, k = 0;
	char content;

	// Multi-threads for many packets. One packet each thread.
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	i = tid + bid * blockDim.x;

	len = packets_len[i];
	for(k = 0; k < len + 1; k++)
	{
		content = packets[MAX_LEN * i + k];
		while(1)
		{
			tmp = acArray[257 * state + ((int)content - 0)];
			if(tmp != 0)
			{
				if(acArray[257 * tmp + 0] != 0)
				{
					contId[j++] = acArray[257 * tmp + 0];
				}
				state = tmp;
				break;
			}
			else
			{
				if(state == 0) break;
				else state = acArray[257 * state + 256];
			}
		}
		if(content == '\0')	break;
	}
	
//	__syncthreads();

	printf("Finish...");
	if(j == 0)
	{	printf("!!!!!!j==0\n");
		result[i] = 0;
	}
	else
	{	printf("!!!!!!j!=0 contId[0]: %d\n", contId[0]);
		result[i] = contId[0];
	}
}

/*__global__ static void match(uint16_t *acArray, AcNodeGPU *contPatt, char *packets, int n, uint16_t *result)
{
	uint16_t contId[10] = {0};
	int tmp;
	int state = 0;
	int i = 0, j = 0, k = 0;

	// Single thread for many packets 
	for(i = 0; i < n; i++)
	{
		for(k = 0; k < LEN; k++)
		{
			char content = packets[LEN * i + k];
			while(1)
			{
				tmp = acArray[257 * state + ((int)content - 0)];
				if(tmp != 0)
				{
					if(acArray[257 * tmp + 0] != 0)
					{
						contId[j++] = acArray[257 * tmp + 0];
					}
					state = tmp;
					break;
				}
				else
				{
					if(state == 0) break;
					else state = acArray[257 * state + 256];
				}
			}
			if(content == '\0')	break;
		}
	}

	__syncthreads();
	for(i = 0; i < 10; i++) result[i] = contId[i];
}*/


/*__device__ uint16_t *acGPU;
__device__ AcNodeGPU *contPatt;
__device__ char *pkt;
__device__ uint16_t *pkt_len;
__device__ uint16_t *res;
__device__ int n;
*/

extern "C"
void gpuinit(RuleSetRoot *rsr, int blockNum, int threadNum, uint16_t **acGPU, AcNodeGPU **contPatt, char **pkt, uint16_t **pkt_len, uint16_t **res)
{
/*	uint16_t *acGPU;
	AcNodeGPU *contPatt;
	char *pkt;
	uint16_t *pkt_len;
	uint16_t *res;*/
	int n = blockNum * threadNum;
	
/*	uint16_t *tmp_acGPU;
	AcNodeGPU *tmp_contPatt;
	char *tmp_pkt;
	uint16_t *tmp_pkt_len;
	uint16_t *tmp_res;
	int tmp_n = blockNum * threadNum;
*/	
	hipMalloc((void **)acGPU, MAX_STATE * 257 * sizeof(uint16_t));
	hipMalloc((void **)contPatt, rsr->nodeNum * sizeof(acGPU));
	hipMalloc((void **)pkt, n * MAX_LEN * sizeof(char));
	hipMalloc((void **)pkt_len, n * sizeof(uint16_t));
	hipMalloc((void **)res, n * sizeof(uint16_t));

	/*hipMalloc((void **)&tmp_acGPU, MAX_STATE * 257 * sizeof(uint16_t));
	hipMalloc((void **)&tmp_contPatt, rsr->nodeNum * sizeof(acGPU));
	hipMalloc((void **)&tmp_pkt, tmp_n * MAX_LEN * sizeof(char));
	hipMalloc((void **)&tmp_pkt_len, tmp_n * sizeof(uint16_t));
	hipMalloc((void **)&tmp_res, tmp_n * sizeof(uint16_t));
	
	hipMemcpyToSymbol(HIP_SYMBOL(acGPU), &tmp_acGPU, MAX_STATE * 257 * sizeof(uint16_t));
	hipMemcpyToSymbol(HIP_SYMBOL(contPatt), &tmp_contPatt, rsr->nodeNum *sizeof(acGPU));
	hipMemcpyToSymbol(HIP_SYMBOL(pkt), &tmp_pkt, tmp_n * MAX_LEN * sizeof(char));
	hipMemcpyToSymbol(HIP_SYMBOL(pkt_len), &tmp_pkt_len, tmp_n * sizeof(uint16_t));
	hipMemcpyToSymbol(HIP_SYMBOL(res), &tmp_res, tmp_n * sizeof(uint16_t));
	hipMemcpyToSymbol(HIP_SYMBOL(n), &tmp_n, sizeof(int));
*/
	hipDeviceProp_t deviceProp;
	int devID = 0;
	hipSetDevice(devID);
	hipGetDeviceProperties(&deviceProp, devID);
	printf("\n\n******GPU Device %s\n", deviceProp.name);
}

extern "C"
void gpufree(int k, uint16_t *acGPU, AcNodeGPU *contPatt, char *pkt, uint16_t *pkt_len, uint16_t *res)
{
/*	uint16_t *acGPU;
	AcNodeGPU *contPatt;
	char *pkt;
	uint16_t *pkt_len;
	uint16_t *res;

	acGPU = *ptr_acGPU;
	contPatt = *ptr_contPatt;
	pkt = *ptr_pkt;
	pkt_len = *ptr_pkt_len;
	res = *ptr_res;
*/
	if(k == 0) 
		printf("######GPU-Free\n");
	hipFree(acGPU);
	hipFree(contPatt);
	hipFree(pkt);
	hipFree(pkt_len);
	hipFree(res);
}

extern "C"
uint16_t *gpumatch(RuleSetRoot *rsr, char *packets, int *packets_len, int blockNum, int threadNum, uint16_t *acGPU, AcNodeGPU *contPatt, char *pkt, uint16_t *pkt_len, uint16_t *res)
{
	int n;
	n = blockNum * threadNum;

/*	uint16_t *acGPU;
	AcNodeGPU *contPatt;
	char *pkt;
	uint16_t *pkt_len;
	uint16_t *res;

	acGPU = *ptr_acGPU;
	contPatt = *ptr_contPatt;
	pkt = *ptr_pkt;
	pkt_len = *ptr_pkt_len;
	res = *ptr_res;
*/
	hipMemcpy(acGPU, rsr->acGPU, MAX_STATE * 257 * sizeof(uint16_t), hipMemcpyHostToDevice);
	hipMemcpy(contPatt, rsr->contPattMatch, rsr->nodeNum * sizeof(AcNodeGPU), hipMemcpyHostToDevice);
	hipMemcpy(pkt, packets, n * MAX_LEN * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(pkt_len, packets_len, n * sizeof(uint16_t), hipMemcpyHostToDevice);

	uint16_t *results;
	results = (uint16_t *)malloc(n * sizeof(uint16_t));
	memset(results, 0, n * sizeof(uint16_t));

	float time_gpu = 0.0;
	hipEvent_t start_gpu, stop_gpu;
	hipEventCreate(&stop_gpu);
	hipEventCreate(&start_gpu);
	hipEventRecord(start_gpu, 0);

	match<<<blockNum, threadNum>>>(acGPU, contPatt, pkt, pkt_len, res);

	hipMemcpy(results, res, n * sizeof(uint16_t), hipMemcpyDeviceToHost);
	
	hipEventRecord(stop_gpu, 0);
	hipEventSynchronize(start_gpu);
	hipEventSynchronize(stop_gpu);
	hipEventElapsedTime(&time_gpu, start_gpu, stop_gpu);

	hipEventDestroy(start_gpu);
	hipEventDestroy(stop_gpu);

	printf("\n\n\n#####gpu time %f(ms)\n", time_gpu);
	printf("######Matching Results:\n");
	int i;
	for(i = 0; i < n; i++)//n; i++)
	{
		if(results[i] != 0) printf("%4d\t%d\n", i, results[i]);
	}
	printf("\n");

	return results;
}
