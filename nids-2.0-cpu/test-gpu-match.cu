#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <rules.h>
#include <assert.h>

#define BEST_BLOCK 15
#define MAX_LEN 1500

__global__ static void match(uint16_t *acArray, AcNodeGPU *contPatt, char *packets, uint16_t *packets_len, uint16_t *result, int *batchSize)
{
	//printf("GPU match\n");
	uint16_t len;
	uint16_t contId[20] = {0};
	int tmp;
	int state = 0;
	int i = 0, j = 0, k = 0, batchNum = 0;
	char content;

	// Multi-threads for many packets. One packet each thread.
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	i = tid + bid * blockDim.x;
	assert(BEST_BLOCK == gridDim.x);

	for(batchNum = 0; i + batchNum * blockDim.x * gridDim.x < batchSize[0]; batchNum++)
	{
	i = i + batchNum * blockDim.x * gridDim.x;
	len = packets_len[i];
	for(k = 0; k < len; k++)
	{
		content = packets[MAX_LEN * i + k];
		while(1)
		{
			tmp = acArray[257 * state + ((int)content - 0)];
			if(tmp != 0)
			{
				if(acArray[257 * tmp + 0] != 0)
				{
					contId[j++] = acArray[257 * tmp + 0];
				}
				state = tmp;
				break;
			}
			else
			{
				if(state == 0) break;
				else state = acArray[257 * state + 256];
			}
		}
		if(content == '\0')	break;
	}
	
	
	//__syncthreads();

//	printf("Finish...");
	if(j == 0)
	{	//printf("!!!!!!j==0\n");
		result[i] = 0;
	}
	else
	{	//printf("!!!!!!j!=0 contId[0]: %d\n", contId[0]);
		result[i] = contId[0];
	}
	}
}

extern "C"
void gpuinit(RuleSetRoot *rsr, int blockNum, int threadNum, int batchSize, uint16_t **acGPU, AcNodeGPU **contPatt, char **pkt, uint16_t **pkt_len, uint16_t **res, int *batch)
{
	int n = batchSize * blockNum * threadNum;
	
	hipMalloc((void **)acGPU, MAX_STATE * 257 * sizeof(uint16_t));
	hipMalloc((void **)contPatt, rsr->nodeNum * sizeof(acGPU));
	hipMalloc((void **)pkt, n * MAX_LEN * sizeof(char));
	hipMalloc((void **)pkt_len, n * sizeof(uint16_t));
	hipMalloc((void **)res, n * sizeof(uint16_t));
	hipMalloc((void **)batch, 1 * sizeof(int));

	hipDeviceProp_t deviceProp;
	int devID = 0;
	hipSetDevice(devID);
	hipGetDeviceProperties(&deviceProp, devID);
	printf("\n\n******GPU Device %s\n", deviceProp.name);
}

extern "C"
void gpufree(int k, uint16_t *acGPU, AcNodeGPU *contPatt, char *pkt, uint16_t *pkt_len, uint16_t *res, int *batch)
{
/*	uint16_t *acGPU;
	AcNodeGPU *contPatt;
	char *pkt;
	uint16_t *pkt_len;
	uint16_t *res;

	acGPU = *ptr_acGPU;
	contPatt = *ptr_contPatt;
	pkt = *ptr_pkt;
	pkt_len = *ptr_pkt_len;
	res = *ptr_res;
*/
	if(k == 0) 
		printf("######GPU-Free\n");
	hipFree(acGPU);
	hipFree(contPatt);
	hipFree(pkt);
	hipFree(pkt_len);
	hipFree(res);
	hipFree(batch);
}

extern "C"
uint16_t *gpumatch(RuleSetRoot *rsr, char *packets, int *packets_len, int blockNum, int threadNum, int batchSize, uint16_t *acGPU, AcNodeGPU *contPatt, char *pkt, uint16_t *pkt_len, uint16_t *res, int *batch)
{
	int host_batch[1];
	int n = batchSize;
	host_batch[0] = batchSize;

/*	uint16_t *acGPU;
	AcNodeGPU *contPatt;
	char *pkt;
	uint16_t *pkt_len;
	uint16_t *res;

	acGPU = *ptr_acGPU;
	contPatt = *ptr_contPatt;
	pkt = *ptr_pkt;
	pkt_len = *ptr_pkt_len;
	res = *ptr_res;

	float time_gpu = 0.0;
	hipEvent_t start_gpu, stop_gpu;
	hipEventCreate(&stop_gpu);
	hipEventCreate(&start_gpu);
	hipEventRecord(start_gpu, 0);
*/

	hipMemcpy(acGPU, rsr->acGPU, MAX_STATE * 257 * sizeof(uint16_t), hipMemcpyHostToDevice);
	hipMemcpy(contPatt, rsr->contPattMatch, rsr->nodeNum * sizeof(AcNodeGPU), hipMemcpyHostToDevice);
	hipMemcpy(pkt, packets, n * MAX_LEN * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpy(pkt_len, packets_len, n * sizeof(uint16_t), hipMemcpyHostToDevice);
	hipMemcpy(batch, host_batch, 1 * sizeof(int), hipMemcpyHostToDevice);

	uint16_t *results;
	results = (uint16_t *)malloc(n * sizeof(uint16_t));
	memset(results, 0, n * sizeof(uint16_t));

	match<<<blockNum, threadNum>>>(acGPU, contPatt, pkt, pkt_len, res, batch);

	hipMemcpy(results, res, n * sizeof(uint16_t), hipMemcpyDeviceToHost);
	
	hipDeviceSynchronize();
/*
	hipEventRecord(stop_gpu, 0);
	hipEventSynchronize(start_gpu);
	hipEventSynchronize(stop_gpu);
	hipEventElapsedTime(&time_gpu, start_gpu, stop_gpu);

	hipEventDestroy(start_gpu);
	hipEventDestroy(stop_gpu);

	printf("\n\n\n######B * T %d\n#####gpu time %f(ms)\n", blockNum * threadNum, time_gpu);
*/
/*	printf("######Matching Results:\n");
	int i;
	for(i = 0; i < n; i++)
	{
		if(results[i] != 0) printf("%4d\t%d\n", i, results[i]);
	}
	printf("\n");
*/
	return results;
}
