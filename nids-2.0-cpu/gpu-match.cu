#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <rules.h>

#define MAX_LEN 1600

__global__ static void match(uint16_t *acArray, char *packets, uint16_t *packets_len, uint16_t *result, int batchSize)
{
	//printf("GPU match\n");
	uint16_t len;
	uint16_t contId[20] = {0};
	int tmp;
	int state = 0;
	int i = 0, j = 0, k = 0, batchNum = 0;
	char content;

	// Multi-threads for many packets. One packet each thread.
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	i = tid + bid * blockDim.x;

	for(batchNum = 0; i + batchNum * blockDim.x * gridDim.x < batchSize; batchNum++)
	{
	i = i + batchNum * blockDim.x * gridDim.x;
	len = packets_len[i];
	for(k = 0; k < len; k++)
	{
		content = packets[MAX_LEN * i + k];
		while(1)
		{
			tmp = acArray[257 * state + ((int)content - 0)];
			if(tmp != 0)
			{
				if(acArray[257 * tmp + 0] != 0)
				{
					contId[j] = acArray[257 * tmp + 0];
					j = 1;
				}
				state = tmp;
				break;
			}
			else
			{
				if(state == 0) break;
				else state = acArray[257 * state + 256];
			}
		}
		if(content == '\0')	break;
	}
	
	
	//__syncthreads();

//	printf("Finish...");
	if(j == 0)
	{	//printf("!!!!!!j==0\n");
		result[i] = 0;
	}
	else
	{	//printf("!!!!!!j!=0 contId[0]: %d\n", contId[0]);
		result[i] = contId[0];
	}
	}
}

extern "C"
void gpuinit(RuleSetRoot *rsr, int blockNum, int threadNum, int batchSize, uint16_t **acGPU, char **pkt, uint16_t **pkt_len, uint16_t **res)
{
	int n;
	n = batchSize;
	
	hipMalloc((void **)acGPU, MAX_STATE * 257 * sizeof(uint16_t));
	hipMalloc((void **)pkt, n * MAX_LEN * sizeof(char));
	hipMalloc((void **)pkt_len, n * sizeof(uint16_t));
	hipMalloc((void **)res, n * sizeof(uint16_t));

	hipMemcpyAsync(*(acGPU), rsr->acGPU, MAX_STATE * 257 * sizeof(uint16_t), hipMemcpyHostToDevice);

	hipDeviceProp_t deviceProp;
	int devID = 0;
	hipSetDevice(devID);
	hipGetDeviceProperties(&deviceProp, devID);
	printf("\n\n******GPU Device %s\n", deviceProp.name);
}

extern "C"
void gpufree(int k, uint16_t *acGPU, char *pkt, uint16_t *pkt_len, uint16_t *res)
{
/*	uint16_t *acGPU;
	AcNodeGPU *contPatt;
	char *pkt;
	uint16_t *pkt_len;
	uint16_t *res;

	acGPU = *ptr_acGPU;
	contPatt = *ptr_contPatt;
	pkt = *ptr_pkt;
	pkt_len = *ptr_pkt_len;
	res = *ptr_res;
*/
	if(k == 0) 
		printf("######GPU-Free\n");
	hipFree(acGPU);
	hipFree(pkt);
	hipFree(pkt_len);
	hipFree(res);
}

extern "C"
uint16_t *gpumatch(RuleSetRoot *rsr, char *packets, int *packets_len, int blockNum, int threadNum, int batchSize, uint16_t *acGPU, char *pkt, uint16_t *pkt_len, uint16_t *res)
{
	int n;
	n = batchSize;

	hipMemcpyAsync(pkt, packets, n * MAX_LEN * sizeof(char), hipMemcpyHostToDevice);
	hipMemcpyAsync(pkt_len, packets_len, n * sizeof(uint16_t), hipMemcpyHostToDevice);

	uint16_t *results;
	results = (uint16_t *)malloc(n * sizeof(uint16_t));
	memset(results, 0, n * sizeof(uint16_t));

	match<<<blockNum, threadNum>>>(acGPU, pkt, pkt_len, res, batchSize);

	hipMemcpyAsync(results, res, n * sizeof(uint16_t), hipMemcpyDeviceToHost);
	
	/*hipError_t err = hipGetLastError();
	if(hipSuccess != err)
	{
		printf("!!!!!!hipGetLastError, %s\n", hipGetErrorString(err));
		//exit(EXIT_FAILURE);
	}*/
//	printf("\n\n\n######B * T %d\n#####gpu time %f(ms)\n", blockNum * threadNum, time_gpu);
/*	printf("######Matching Results:\n");
	int i;
	for(i = 0; i < n; i++)
	{
		if(results[i] != 0) printf("%4d\t%d\n", i, results[i]);
	}
	printf("\n");
*/
	return results;
}
